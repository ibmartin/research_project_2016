#include "hip/hip_runtime.h"
#ifndef _GPU_PROC_HOST_CU_
#define _GPU_PROC_HOST_CU_

#include <atomic>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "GpuProcKernel.cu"

#define FILTER_SIZE			3
#define M_PI				3.14159265358979323846  /* pi */
#define IMG_CHUNK			3110400	/* (1920 x 1080 x 3) / 2 */
#define THREADS_PER_BLOCK	256
#define MEM_CAP				32768 //32 KB as a power of 2
//#define MEM_CAP				16384 //32 KB as a power of 2

void cudaRgb2Gray(unsigned char* input, unsigned char* output, int srcRows, int srcCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);

	//int destN = min(1920 * 1080, srcRows * srcCols);
	for (int step = 0; step < rounds; step++){
		int destN = fmin(chunkRows * srcCols, srcRows * srcCols - offset);
		if (destN <= 0)
			break;

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));
		rgb2GrayKernel <<<blocks, threadsPerBlock >>> (deviceDestData, deviceSrcData, srcRows, srcCols, chunkRows, offset);
		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}

	hipFree(deviceSrcData);
}

void cudafRgb2Gray(unsigned char* input, float* output, int srcRows, int srcCols){
	unsigned char* deviceSrcData;
	float* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	//int srcN = 3 * srcRows * srcCols;
	int datasize1 = 3 * sizeof(unsigned char);
	int datasize2 = sizeof(float);

	int chunk_size = MEM_CAP / (datasize1 + datasize2);
	int rounds = ceil((srcRows * srcCols) / (float)chunk_size);
	//hipMalloc(&deviceSrcData, srcN*sizeof(float));
	//hipMemcpy(deviceSrcData, input, srcN*sizeof(float), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	//int rounds = ceil(srcRows / (double)chunkRows);
	//int rounds = steps;

	for (int step = 0; step < rounds; step++){
		int remainder = fmin(chunk_size, (srcRows * srcCols) - (step * chunk_size) );
		int srcN = remainder * datasize1;
		int destN = remainder * datasize2;
		if (destN <= 0)
			break;

		blocks = ((destN / datasize2) + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceSrcData, srcN);
		hipMemcpy(deviceSrcData, input + (step * chunk_size * 3), srcN, hipMemcpyHostToDevice);

		hipMalloc(&deviceDestData, destN);
		frgb2GrayKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, srcRows, srcCols, chunkRows, 0);
		hipMemcpy(output + offset, deviceDestData, destN, hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += remainder;
	}
}

void cudaReverse(unsigned char* input, unsigned char* output, int srcRows, int srcCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);
	//printf("Rounds: %d \n", rounds);

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - (offset * 3));
		if (destN <= 0){
			//printf("Broken!\n");
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));

		reverseKernel <<<blocks, threadsPerBlock >>>(deviceDestData, deviceSrcData, srcN, chunkRows, offset);
		hipMemcpy(output + (3 * offset), deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN / 3;
	}


	hipFree(deviceSrcData);

}

void cudaGammaCorrection(unsigned char* input, unsigned char* output, double gamma, int srcRows, int srcCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));

		gammaCorrectionKernel <<<blocks, threadsPerBlock >>> (deviceDestData, deviceSrcData, srcRows, srcCols, gamma, chunkRows, offset);

		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}

	hipFree(deviceSrcData);

}

void cudaDirectResize(unsigned char* input, unsigned char* output, int srcRows, int srcCols, int destRows, int destCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / destCols;
	if (chunkRows == 0){
		chunkRows = destRows;
	}
	int rounds = ceil(destRows / (double)chunkRows);

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * destCols, 3 * destRows * destCols - offset);
		if (destN <= 0){
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));

		directResizeKernel <<<blocks, threadsPerBlock >>> (deviceDestData, deviceSrcData, srcRows, srcCols, destRows, destCols, chunkRows, offset);
		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}
	hipFree(deviceSrcData);
}

void cudafDirectResize(float* input, float* output, int srcRows, int srcCols, int destRows, int destCols){
	float* deviceSrcData;
	float* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;

	int datasize = sizeof(float);

	float ratio = (float)(destRows * destCols) / (float)(srcRows * srcCols);
	ratio = ((1 / ratio) + 1) * datasize;

	int remainder = destRows * destCols;
	float rRow = (float)srcRows / destRows;
	float rCol = (float)srcCols / destCols;

	int pixels = ceil(MEM_CAP / ratio);
	int sentinel = 1;

	while (remainder > 0){

		//printf("Pixels: %d\n", pixels);

		int pix_begin = (destRows * destCols) - remainder;
		int pix_end = min(destRows * destCols - 1, pix_begin + pixels - 1);
		//printf("Begin: %d, End: %d\n", pix_begin, pix_end);

		int destN = (pix_end - pix_begin + 1) * datasize;
		blocks = ((destN / datasize) + threadsPerBlock - 1) / threadsPerBlock;

		int sRow = ((float)pix_begin / destCols) * rRow;
		int sCol = (pix_begin % destCols) * rCol;
		int src_begin = (sRow * srcCols + sCol);

		sRow = ((float)pix_end / destCols) * rRow;	// (3531 / 1600) * (480 / 1200) = 0.88275
		sCol = (pix_end % destCols) * rCol;			// (3531 % 1600) * (640 / 1600) = 132.4
		int src_end = ((sRow + 1) * srcCols + sCol);

		//printf("srcRows: %d, srcCols: %d, total: %d\n", srcRows, srcCols, srcRows * srcCols);

		//printf("SBegin: %d, SEnd: %d\n", src_begin, src_end);

		int srcN = (src_end - src_begin + 1) * datasize;

		hipMalloc(&deviceSrcData, srcN);
		hipMalloc(&deviceDestData, destN);
		hipMemcpy(deviceSrcData, input + src_begin, srcN, hipMemcpyHostToDevice);

		fdirectResizeKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, srcRows, srcCols, destRows, destCols, chunkRows, offset);

		hipMemcpy(output + pix_begin, deviceDestData, destN, hipMemcpyDeviceToHost);
		
		hipFree(deviceDestData);
		hipFree(deviceSrcData);

		remainder -= pix_end - pix_begin + 1;
		sentinel -= 1;

	}

	


}

void cudaLinearResize(unsigned char* input, unsigned char* output, int srcRows, int srcCols, int destRows, int destCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / destCols;
	if (chunkRows == 0){
		chunkRows = destRows;
	}
	int rounds = ceil(destRows / (double)chunkRows);

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * destCols, 3 * destRows * destCols - offset);
		if (destN <= 0){
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));

		linearResizeKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, srcRows, srcCols, destRows, destCols, chunkRows, offset);

		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}

	hipFree(deviceSrcData);
}

void cudaGaussianFilter(unsigned char* input, unsigned char* output, double* gKernel, int srcRows, int srcCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	double* deviceFilter;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMalloc(&deviceFilter, (2 * FILTER_SIZE + 1) * (2 * FILTER_SIZE + 1) * sizeof(double));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(deviceFilter, gKernel, (2 * FILTER_SIZE + 1)*(2 * FILTER_SIZE + 1)*sizeof(double), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));

		gaussianFilterKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, deviceFilter, FILTER_SIZE, srcRows, srcCols, chunkRows, offset);

		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}

	hipFree(deviceSrcData);
	hipFree(deviceFilter);
}

void fCudaGaussianFilter(float* input, float* output, double* gKernel, int srcRows, int srcCols){
	float* deviceSrcData;
	float* deviceDestData;
	double* deviceFilter;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int srcN = 3 * srcRows * srcCols;

	hipMalloc(&deviceSrcData, srcN*sizeof(float));
	hipMalloc(&deviceFilter, (2 * FILTER_SIZE + 1) * (2 * FILTER_SIZE + 1) * sizeof(double));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceFilter, gKernel, (2 * FILTER_SIZE + 1)*(2 * FILTER_SIZE + 1)*sizeof(double), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);

	int offset = 0;
	for (int step = 0; step < rounds; step++){
		printf("Step\n");
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(float));
		//hipMalloc(&deviceSrcData, destN*sizeof(float));

		//hipMemcpy(deviceSrcData, input + offset, destN*sizeof(float), hipMemcpyHostToDevice);

		//fGaussianFilterKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, deviceFilter, FILTER_SIZE, srcRows, srcCols, chunkRows, offset);

		//hipMemcpy(output + offset, deviceDestData, destN*sizeof(float), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);
		//hipFree(deviceSrcData);

		offset += destN;
	}

	hipFree(deviceFilter);
	hipFree(deviceSrcData);

}

void cudaSobelFilter(unsigned char* input, unsigned char* output, int srcRows, int srcCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	short* deviceTempData;
	int* deviceSobel_x;
	int* deviceSobel_y;
	int srcN = 3 * srcRows * srcCols;
	double* deviceRangeMin;
	double* deviceRangeMax;
	double rangeMin[1] = { 0.0 };
	double rangeMax[1] = { 0.0 };
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;

	int sobel_x[9], sobel_y[9];

	sobel_x[0] = -1; sobel_x[1] = 0; sobel_x[2] = 1;
	sobel_x[3] = -2; sobel_x[4] = 0; sobel_x[5] = 2;
	sobel_x[6] = -1; sobel_x[7] = 0; sobel_x[8] = 1;

	sobel_y[0] = -1; sobel_y[1] = -2; sobel_y[2] = -1;
	sobel_y[3] = 0; sobel_y[4] = 0; sobel_y[5] = 0;
	sobel_y[6] = 1; sobel_y[7] = 2; sobel_y[8] = 1;

	//int threadsPerBlock = 512;
	//blocks = (srcN + threadsPerBlock - 1) / threadsPerBlock;
	//printf("Blocks: %d\n", blocks);

	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMalloc(&deviceSobel_x, 9 * sizeof(int));
	hipMalloc(&deviceSobel_y, 9 * sizeof(int));
	hipMalloc(&deviceRangeMin, sizeof(double));
	hipMalloc(&deviceRangeMax, sizeof(double));

	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(deviceSobel_x, sobel_x, 9 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceSobel_y, sobel_y, 9 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceRangeMin, rangeMin, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(deviceRangeMax, rangeMax, sizeof(double), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);

	short* temp_data = new short[3 * srcRows * srcCols];

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}
		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceTempData, destN*sizeof(short));

		sobelGradientKernel << <blocks, threadsPerBlock >> > (deviceTempData, deviceSrcData, deviceSobel_x, deviceSobel_y, deviceRangeMin, deviceRangeMax, srcRows, srcCols, offset);

		hipMemcpy(temp_data + offset, deviceTempData, destN*sizeof(short), hipMemcpyDeviceToHost);
		hipFree(deviceTempData);

		offset += destN;
	}
	hipFree(deviceSrcData);
	hipFree(deviceSobel_x);
	hipFree(deviceSobel_y);
	hipFree(deviceRangeMin);
	hipFree(deviceRangeMax);
	//printf("Works!\n");

	//hipMemcpy(rangeMin, deviceRangeMin, sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(rangeMax, deviceRangeMax, sizeof(double), hipMemcpyDeviceToHost);

	//printf("Host temp data done");

	for (int i = 0; i < srcRows; i++){
		for (int j = 0; j < srcCols; j++){
			for (int color = 0; color < 3; color++){
				double value = temp_data[3 * (i * srcCols + j) + color];;
				rangeMin[0] = std::fmin(value, rangeMin[0]);
				rangeMax[0] = std::fmax(value, rangeMax[0]);
			}
		}
	}


	//printf("Got here!\n");
	//output = (uchar*)temp_data;
	//return;

	//printf("Range Min: %f, Range Max: %f \n", rangeMin[0], rangeMax[0]);

	//blocks = (srcN + threadsPerBlock - 1) / threadsPerBlock;
	offset = 0;

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}
		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceTempData, destN*sizeof(short));
		hipMemcpy(deviceTempData, temp_data + offset, destN*sizeof(short), hipMemcpyHostToDevice);

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));
		sobelRangeKernel << <blocks, threadsPerBlock >> >(deviceDestData, deviceTempData, rangeMin[0], rangeMax[0], 20, 60, offset);

		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);

		hipFree(deviceDestData);
		hipFree(deviceTempData);
		offset += destN;
	}

	//uchar minThresh = 20;
	//uchar maxThresh = 60;

	/*for (int i = 0; i < srcRows; i++){
	for (int j = 0; j < srcCols; j++){
	for (int color = 0; color < 3; color++){
	int idx = 3 * (i * srcCols + j) + color;

	int value = 255 * ((temp_data[idx] + rangeMin[0])/(rangeMax[0] - rangeMin[0]));
	if (value >= maxThresh){
	value = 255;
	}
	else if (value < minThresh){
	value = 0;
	}
	output[idx] = value;
	}
	}
	}*/


	//hipFree(deviceTempData);
	//hipFree(deviceDestData);

	delete[] temp_data;
}

void cudaKMeans(unsigned char* input, unsigned char* output, int srcRows, int srcCols, int k_means){
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;

	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	float* device_k_colors;
	int* device_k_count;
	//int* device_hits;
	unsigned char* device_k_index;
	bool* device_convergence;

	float* k_colors = new float[k_means * 3];
	unsigned char* k_index = new unsigned char[srcRows * srcCols];
	int* k_count = new int[k_means];
	//* hits = new int[k_means];

	int srcN = srcRows * srcCols * 3;

	for (int pix = 0; pix < k_means; pix++){
		int i = rand() % srcRows;
		int j = rand() % srcCols;
		for (int color = 0; color < 3; color++){
			k_colors[3 * pix + color] = input[3 * (i * srcCols + j) + color];
		}
	}
	hipMalloc(&device_k_colors, (3 * k_means)*sizeof(float));
	hipMemcpy(device_k_colors, k_colors, 3 * k_means *sizeof(float), hipMemcpyHostToDevice);

	//printf("=== START ===\n");
	//for (int group = 0; group < k_means; group++){
	//printf("Color Group %d: R=%f, G=%f, B=%f \n", group + 1, k_colors[3 * group + 2], k_colors[3 * group + 1], k_colors[3 * group]);
	//}

	bool convergence[1] = { false };

	for (int k = 0; k < srcRows * srcCols; k++){
		k_index[k] = 0;
	}

	chunkRows = (IMG_CHUNK * 0.5) / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}

	hipMalloc(&device_k_count, (k_means)*sizeof(int));
	//hipMalloc(&device_hits, (k_means)*sizeof(int));
	//hipMalloc(&device_k_colors, (3 * k_means)*sizeof(float));
	hipMalloc(&device_convergence, sizeof(bool));

	//hipMalloc(&device_k_index, (srcRows * srcCols)*sizeof(uchar));
	//hipMemcpy(device_k_index, k_index, srcRows * srcCols *sizeof(uchar), hipMemcpyHostToDevice);
	int count = 0;

	while (!convergence[0]){
		convergence[0] = true;
		hipMemcpy(device_convergence, convergence, sizeof(bool), hipMemcpyHostToDevice);
		for (int k = 0; k < k_means; k++){
			k_count[k] = 0;
			//hits[k] = 0;
		}
		hipMemcpy(device_k_count, k_count, k_means * sizeof(int), hipMemcpyHostToDevice);
		//hipMemcpy(device_hits, hits, k_means * sizeof(int), hipMemcpyHostToDevice);
		//printf("Count: %d\n",count);

		int rounds = ceil(srcRows / (float)chunkRows);

		offset = 0;
		for (int step = 0; step < rounds; step++){
			int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
			if (destN <= 0){
				break;
			}
			blocks = ((destN / 3) + threadsPerBlock - 1) / threadsPerBlock;

			hipMalloc(&deviceSrcData, destN*sizeof(unsigned char));
			hipMemcpy(deviceSrcData, input + offset, destN*sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMalloc(&device_k_index, destN*sizeof(unsigned char) / 3);
			hipMemcpy(device_k_index, k_index + (offset / 3), destN*sizeof(unsigned char) / 3, hipMemcpyHostToDevice);

			//kernel
			//kMeansCountingKernel << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_hits, device_k_colors, device_convergence, k_means, srcRows, srcCols,count);
			kMeansCountingKernel << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_k_colors, device_convergence, k_means, srcRows, srcCols);

			hipMemcpy(k_index + (offset / 3), device_k_index, destN*sizeof(unsigned char) / 3, hipMemcpyDeviceToHost);

			hipFree(deviceSrcData);
			hipFree(device_k_index);
			offset += destN;
		}
		hipMemcpy(k_count, device_k_count, (k_means)*sizeof(int), hipMemcpyDeviceToHost);
		//hipMemcpy(hits, device_hits, (k_means)*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(convergence, device_convergence, sizeof(bool), hipMemcpyDeviceToHost);

		//printf("Group Count, step %d::\n",count);
		//for (int i = 0; i < k_means; i++){
		//	printf("Group %d: %d\n",i,k_count[i]);
		//}

		//convergence[0] = true;	//Stopper

		//if (convergence[0])		//RE-ENABLE WHEN ACTUALLY USING
		//break;

		if (count == 200){
			//printf("Stopped at %d!\n",count);
			break;
		}
		count++;

		for (int k = 0; k < 3 * k_means; k++){
			k_colors[k] = 0;
		}
		hipMemcpy(device_k_colors, k_colors, 3 * k_means *sizeof(float), hipMemcpyHostToDevice);

		rounds = ceil(srcRows / (float)chunkRows);

		offset = 0;
		for (int step = 0; step < rounds; step++){
			int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
			if (destN <= 0){
				break;
			}
			blocks = ((destN / 3) + threadsPerBlock - 1) / threadsPerBlock;

			hipMalloc(&deviceSrcData, destN*sizeof(unsigned char));
			hipMemcpy(deviceSrcData, input + offset, destN*sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMalloc(&device_k_index, destN*sizeof(unsigned char) / 3);
			hipMemcpy(device_k_index, k_index + (offset / 3), destN*sizeof(unsigned char) / 3, hipMemcpyHostToDevice);

			kMeansGroupAdjustKernel << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_k_colors, k_means, srcRows, srcCols);

			hipFree(deviceSrcData);
			hipFree(device_k_index);

			offset += destN;
		}
		hipMemcpy(k_colors, device_k_colors, 3 * k_means * sizeof(float), hipMemcpyDeviceToHost);
		//kernel
	}

	int rounds = ceil(srcRows / (float)chunkRows);
	offset = 0;
	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}
		blocks = ((destN / 3) + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));
		hipMalloc(&device_k_index, destN*sizeof(unsigned char) / 3);
		hipMemcpy(device_k_index, k_index + (offset / 3), destN*sizeof(unsigned char) / 3, hipMemcpyHostToDevice);

		//kernel
		kMeansOutputKernel << <blocks, threadsPerBlock >> > (deviceDestData, device_k_index, device_k_colors, srcRows, srcCols);
		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);

		hipFree(deviceDestData);
		hipFree(device_k_index);

		offset += destN;
	}

	//printf("Count: %d\n", count);

	hipFree(device_k_colors);
	hipFree(device_k_count);
	hipFree(device_convergence);

	delete[] k_colors;
	delete[] k_index;
	delete[] k_count;

}

void cudaKMeansOld(unsigned char* input, unsigned char* output, int srcRows, int srcCols, int k_means){
	int threadsPerBlock = 512;
	int blocks = ((srcRows * srcCols) + threadsPerBlock - 1) / threadsPerBlock;
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	float* device_k_colors;
	int* device_k_count;
	unsigned char* device_k_index;
	bool* device_convergence;
	//int srcN = 3 * srcRows * srcCols;
	int srcN = fmin(3 * 1920 * 1080, 3 * srcRows * srcCols);
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMalloc(&deviceDestData, srcN*sizeof(unsigned char));
	hipMalloc(&device_k_colors, (3 * k_means)*sizeof(float));
	hipMalloc(&device_k_index, (srcRows * srcCols)*sizeof(unsigned char));
	hipMalloc(&device_k_count, (k_means)*sizeof(int));
	hipMalloc(&device_convergence, sizeof(bool));

	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	float* k_colors = new float[3 * k_means];
	unsigned char* k_index = new unsigned char[srcRows * srcCols];
	int* k_count = new int[k_means];

	for (int pix = 0; pix < k_means; pix++){
		int i = rand() % srcRows;
		int j = rand() % srcCols;
		for (int color = 0; color < 3; color++){
			k_colors[3 * pix + color] = input[3 * (i * srcCols + j) + color];
		}

	}
	hipMemcpy(device_k_colors, k_colors, 3 * k_means *sizeof(float), hipMemcpyHostToDevice);

	printf("=== START ===\n");
	for (int group = 0; group < k_means; group++){
		printf("Color Group %d: R=%f, G=%f, B=%f \n", group + 1, k_colors[3 * group + 2], k_colors[3 * group + 1], k_colors[3 * group]);
	}

	bool convergence[1] = { false };

	for (int k = 0; k < srcRows * srcCols; k++){
		k_index[k] = 0;
	}
	hipMemcpy(device_k_index, k_index, srcRows * srcCols *sizeof(unsigned char), hipMemcpyHostToDevice);
	int count = 0;

	while (!convergence[0]){
		convergence[0] = true;
		hipMemcpy(device_convergence, convergence, sizeof(bool), hipMemcpyHostToDevice);
		for (int k = 0; k < k_means; k++){
			k_count[k] = 0;
		}
		hipMemcpy(device_k_count, k_count, k_means * sizeof(int), hipMemcpyHostToDevice);

		kMeansCountingKernelOld << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_k_colors, device_convergence, k_means, srcRows, srcCols);
		hipMemcpy(k_index, device_k_index, (srcRows*srcCols)*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipMemcpy(k_count, device_k_count, (k_means)*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(convergence, device_convergence, sizeof(bool), hipMemcpyDeviceToHost);

		if (count == 400){
			printf("Stopped at 400!\n");
			break;
		}
		count++;
		//printf("Bogey::\n");
		if (convergence[0])
			break;
		for (int k = 0; k < 3 * k_means; k++){
			k_colors[k] = 0;
		}
		hipMemcpy(device_k_colors, k_colors, 3 * k_means *sizeof(float), hipMemcpyHostToDevice);

		//kMeansGroupAdjustKernel<<<blocks,threadsPerBlock>>> (deviceSrcData,device_k_index,device_k_count,device_k_colors,k_means,srcRows,srcCols);
		hipMemcpy(k_colors, device_k_colors, 3 * k_means * sizeof(float), hipMemcpyDeviceToHost);

	}
	hipMemcpy(device_k_colors, k_colors, 3 * k_means *sizeof(float), hipMemcpyHostToDevice);


	printf("=== END ===\n");
	//for (int group = 0; group < k_means; group++){
	//printf("Color Group %d: R=%f, G=%f, B=%f \n", group + 1, k_colors[3 * group + 2], k_colors[3 * group + 1], k_colors[3 * group]);
	//}
	kMeansOutputKernel << <blocks, threadsPerBlock >> > (deviceDestData, device_k_index, device_k_colors, srcRows, srcCols);
	hipMemcpy(output, deviceDestData, srcN*sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(deviceSrcData);
	hipFree(deviceDestData);
	hipFree(device_k_colors);
	hipFree(device_k_index);
	hipFree(device_k_count);
	hipFree(device_convergence);

	delete[] k_colors;
	delete[] k_index;
	delete[] k_count;

}

void cudaMySiftDOG(float* current, float* next, float* dog, int curRows, int curCols){
	float* deviceCurrData;
	float* deviceNextData;
	float* deviceDogData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int datasize = sizeof(float);

	int pixels = MEM_CAP / (3 * datasize);

	int remainder = curRows * curCols;

	while (remainder > 0){

		int pix_begin = (curRows * curCols) - remainder;
		int pix_end = min(curRows * curCols - 1, pix_begin + pixels - 1);

		int destN = (pix_end - pix_begin + 1) * datasize;
		int blocks = ((destN / datasize) + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceCurrData, destN);
		hipMalloc(&deviceNextData, destN);
		hipMalloc(&deviceDogData, destN);
		hipMemcpy(deviceCurrData, current + pix_begin, destN, hipMemcpyHostToDevice);
		hipMemcpy(deviceNextData, next + pix_begin, destN, hipMemcpyHostToDevice);

		mySiftDOGKernel << <blocks, threadsPerBlock >> >(deviceCurrData, deviceNextData, deviceDogData);

		hipMemcpy(dog + pix_begin, deviceDogData, destN, hipMemcpyDeviceToHost);

		hipFree(deviceCurrData);
		hipFree(deviceNextData);
		hipFree(deviceDogData);

		remainder -= pixels;
	}

}

#endif