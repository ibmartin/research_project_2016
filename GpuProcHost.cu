#include "hip/hip_runtime.h"
#ifndef _GPU_PROC_HOST_CU_
#define _GPU_PROC_HOST_CU_

#include <atomic>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "GpuProcKernel.cu"

#define FILTER_SIZE			3
#define M_PI				3.14159265358979323846  /* pi */
#define IMG_CHUNK			3110400	/* (1920 x 1080 x 3) / 2 */
#define THREADS_PER_BLOCK	256
//#define MEM_CAP				131072	// (2 ^ 17)
#define MEM_CAP				65536 //64 KB as a power of 2 (2 ^ 16)
//#define MEM_CAP				32768 //32 KB as a power of 2 (2 ^ 15)
//#define MEM_CAP				16384 //16 KB as a power of 2 (2 ^ 14)

#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

int FIXED = 16;
int ONE = 1 << FIXED;

void cudaLagSetup(){
	get_nvtAttrib("Cuda Setup", 0xFFFFFFFF);
	unsigned char* nothing;
	hipMalloc(&nothing, sizeof(unsigned char));
	hipFree(nothing);
	nvtxRangePop();
}

void cudaRgb2Gray(unsigned char* input, unsigned char* output, int srcRows, int srcCols){
	get_nvtAttrib("rgb2Gray CPU", 0xFF222222);
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);

	//int destN = min(1920 * 1080, srcRows * srcCols);
	for (int step = 0; step < rounds; step++){
		int destN = fmin(chunkRows * srcCols, srcRows * srcCols - offset);
		if (destN <= 0)
			break;

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));
		rgb2GrayKernel <<<blocks, threadsPerBlock >>> (deviceDestData, deviceSrcData, srcRows, srcCols, chunkRows, offset);
		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}

	hipFree(deviceSrcData);
	nvtxRangePop();
}

void cudafRgb2Gray(unsigned char* input, float* output, int srcRows, int srcCols){
	get_nvtAttrib("Setup", 0xFF0000FF);
	unsigned char* deviceSrcData;
	float* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	//int srcN = 3 * srcRows * srcCols;
	int datasize1 = 3 * sizeof(unsigned char);
	int datasize2 = sizeof(float);

	int chunk_size = MEM_CAP / (datasize1 + datasize2);
	int rounds = ceil((srcRows * srcCols) / (float)chunk_size);
	//hipMalloc(&deviceSrcData, srcN*sizeof(float));
	//hipMemcpy(deviceSrcData, input, srcN*sizeof(float), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	//int rounds = ceil(srcRows / (double)chunkRows);
	//int rounds = steps;
	nvtxRangePop();//Setup

	get_nvtAttrib("K Blocks " + std::to_string(rounds), 0xFF008800);
	for (int step = 0; step < rounds; step++){
		get_nvtAttrib("Pre-K", 0xFFFF0000);
		get_nvtAttrib("Vars", 0xFF880000);
		int remainder = fmin(chunk_size, (srcRows * srcCols) - (step * chunk_size) );
		int srcN = remainder * datasize1;
		int destN = remainder * datasize2;
		if (destN <= 0)
			break;

		blocks = ((destN / datasize2) + threadsPerBlock - 1) / threadsPerBlock;
		nvtxRangePop();

		get_nvtAttrib("Malloc", 0xFF880000);
		hipMalloc(&deviceSrcData, srcN);
		hipMalloc(&deviceDestData, destN);
		nvtxRangePop();

		get_nvtAttrib("Memcpy", 0xFF880000);
		hipMemcpy(deviceSrcData, input + (step * chunk_size * 3), srcN, hipMemcpyHostToDevice);
		nvtxRangePop();
		
		nvtxRangePop();

		get_nvtAttrib("Kern", 0xFF00FF00);
		frgb2GrayKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, srcRows, srcCols, chunkRows, 0);
		hipDeviceSynchronize();
		nvtxRangePop();

		get_nvtAttrib("Post-K", 0xFF0000FF);
		hipMemcpy(output + offset, deviceDestData, destN, hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += remainder;
		nvtxRangePop();
	}
	nvtxRangePop();
}

void cudaReverse(unsigned char* input, unsigned char* output, int srcRows, int srcCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);
	//printf("Rounds: %d \n", rounds);

	for (int step = 0; step < rounds; step++){
		int destN = fmin(chunkRows * srcCols, srcRows * srcCols - (offset));
		if (destN <= 0){
			//printf("Broken!\n");
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN * 3 * sizeof(unsigned char));

		reverseKernel <<<blocks, threadsPerBlock >>>(deviceDestData, deviceSrcData, srcN, chunkRows, offset);
		hipMemcpy(output + (3 * offset), deviceDestData, destN * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}


	hipFree(deviceSrcData);

}

void cudaGammaCorrection(unsigned char* input, unsigned char* output, double gamma, int srcRows, int srcCols){
	get_nvtAttrib("Setup Inner", 0xFF000088);
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);
	nvtxRangePop();

	get_nvtAttrib("Work Loop: " + std::to_string(srcRows * srcCols / rounds), 0xFF888888);
	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));

		get_nvtAttrib("Kernel", 0xFF00FF00);
		gammaCorrectionKernel <<<blocks, threadsPerBlock >>> (deviceDestData, deviceSrcData, srcRows, srcCols, gamma, chunkRows, offset);
		nvtxRangePop();

		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}
	nvtxRangePop();

	hipFree(deviceSrcData);

}

void cudaDirectResize(unsigned char* input, unsigned char* output, int srcRows, int srcCols, int destRows, int destCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / destCols;
	if (chunkRows == 0){
		chunkRows = destRows;
	}
	int rounds = ceil(destRows / (double)chunkRows);

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * destCols, 3 * destRows * destCols - offset);
		if (destN <= 0){
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));

		directResizeKernel <<<blocks, threadsPerBlock >>> (deviceDestData, deviceSrcData, srcRows, srcCols, destRows, destCols, chunkRows, offset);
		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}
	hipFree(deviceSrcData);
}

void cudafDirectResize(float* input, float* output, int srcRows, int srcCols, int destRows, int destCols){	//
	float* deviceSrcData;
	float* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;

	int datasize = sizeof(float);

	float ratio = (float)(destRows * destCols) / (float)(srcRows * srcCols);
	ratio = ((1 / ratio) + 1) * datasize;

	int remainder = destRows * destCols;
	float rRow = (float)srcRows / destRows;
	float rCol = (float)srcCols / destCols;

	int pixels = ceil(MEM_CAP / ratio);
	int sentinel = 1;

	while (remainder > 0){

		//printf("Pixels: %d\n", pixels);

		int pix_begin = (destRows * destCols) - remainder;
		int pix_end = min(destRows * destCols - 1, pix_begin + pixels - 1);
		//printf("Begin: %d, End: %d\n", pix_begin, pix_end);

		int destN = (pix_end - pix_begin + 1) * datasize;
		blocks = ((destN / datasize) + threadsPerBlock - 1) / threadsPerBlock;

		int sRow = (pix_begin / destCols) * rRow;
		//int sCol = (pix_begin % destCols) * rCol;
		int sCol = 0;
		int src_begin = (sRow * srcCols + sCol);

		sRow = ((float)pix_end / destCols) * rRow;	// (3531 / 1600) * (480 / 1200) = 0.88275
		sCol = (pix_end % destCols) * rCol;			// (3531 % 1600) * (640 / 1600) = 132.4
		int src_end = min(srcRows * srcCols - 1, ((sRow + 1) * srcCols + sCol));

		//printf("srcRows: %d, srcCols: %d, total: %d\n", srcRows, srcCols, srcRows * srcCols);

		//printf("SBegin: %d, SEnd: %d\n", src_begin, src_end);
		/*if (pix_begin / destCols <= 11 && pix_end / destCols >= 11){
			printf("Pix Begin: %d, %d\n", pix_begin / destCols, pix_begin % destCols);
			printf("  sRow: %f, sCol: %f\n", (pix_begin / destCols) * rRow, (pix_begin % destCols) * rCol);
			printf("Pix End  : %d, %d\n", pix_end / destCols, pix_end % destCols);
			printf("Src Begin: %d, %d\n", src_begin / srcCols, src_begin % srcCols);
			printf("Src End  : %d, %d\n", src_end / srcCols, src_end % srcCols);
		}*/

		int srcN = (src_end - src_begin + 1) * datasize;

		hipMalloc(&deviceSrcData, srcN);
		hipMalloc(&deviceDestData, destN);
		//printf("Size: %d, src_begin: %d, src_end: %d\n", srcRows * srcCols, src_begin, src_end);
		hipMemcpy(deviceSrcData, input + src_begin, srcN, hipMemcpyHostToDevice);

		fdirectResizeKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, srcRows, srcCols, destRows, destCols, pix_begin, src_begin);

		hipMemcpy(output + pix_begin, deviceDestData, destN, hipMemcpyDeviceToHost);
		
		hipFree(deviceDestData);
		hipFree(deviceSrcData);

		remainder -= pix_end - pix_begin + 1;
		sentinel -= 1;

	}
	return;
}

void cudaLinearResize(unsigned char* input, unsigned char* output, int srcRows, int srcCols, int destRows, int destCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / destCols;
	if (chunkRows == 0){
		chunkRows = destRows;
	}
	int rounds = ceil(destRows / (double)chunkRows);

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * destCols, 3 * destRows * destCols - offset);
		if (destN <= 0){
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));

		linearResizeKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, srcRows, srcCols, destRows, destCols, chunkRows, offset);

		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}

	hipFree(deviceSrcData);
}

void cudaGaussianFilter(unsigned char* input, unsigned char* output, double* gKernel, int srcRows, int srcCols){
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	double* deviceFilter;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;
	int srcN = 3 * srcRows * srcCols;
	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMalloc(&deviceFilter, (2 * FILTER_SIZE + 1) * (2 * FILTER_SIZE + 1) * sizeof(double));
	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(deviceFilter, gKernel, (2 * FILTER_SIZE + 1)*(2 * FILTER_SIZE + 1)*sizeof(double), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);

	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}

		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));

		gaussianFilterKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, deviceFilter, FILTER_SIZE, srcRows, srcCols, chunkRows, offset);

		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipFree(deviceDestData);

		offset += destN;
	}

	hipFree(deviceSrcData);
	hipFree(deviceFilter);
}

void cudafGaussianFilter(float* input, float* output, double* gKernel, int srcRows, int srcCols){
	get_nvtAttrib("Setup", 0xFF0000FF);
	float* deviceSrcData;
	float* deviceDestData;
	double* deviceFilter;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int datasize = sizeof(float);
	int srcN = srcRows * srcCols;

	hipMalloc(&deviceFilter, (2 * FILTER_SIZE + 1) * (2 * FILTER_SIZE + 1) * sizeof(double));
	hipMemcpy(deviceFilter, gKernel, (2 * FILTER_SIZE + 1) * (2 * FILTER_SIZE + 1) * sizeof(double), hipMemcpyHostToDevice);

	int remainder = srcRows * srcCols;
	int pixels = MEM_CAP / 4.0 * datasize;

	chunkRows = MEM_CAP / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);

	int offset = 0;
	nvtxRangePop();//Setup
	//for (int step = 0; step < rounds; step++){
	get_nvtAttrib("K Blocks " + std::to_string(rounds), 0xFF008800);
	while (remainder > 0){
		get_nvtAttrib("Pre-K", 0xFFFF0000);
		int pix_begin = (srcRows * srcCols) - remainder;
		int pix_end = min(srcRows * srcCols - 1, pix_begin + pixels - 1);

		//int destN = fmin(pixels, remainder);
		int destN = (pix_end - pix_begin + 1) * datasize;
		if (destN <= 0){
			break;
		}

		blocks = ((destN / datasize) + threadsPerBlock - 1) / threadsPerBlock;

		int src_begin = max((pix_begin - FILTER_SIZE) - (FILTER_SIZE * srcCols), 0);
		int src_end = min((pix_end + FILTER_SIZE) + (FILTER_SIZE * srcCols), srcRows * srcCols - 1);
		int srcN = (src_end - src_begin + 1) * datasize;

		get_nvtAttrib("Malloc", 0xFFFF0000);
		hipMalloc(&deviceDestData, destN);
		hipMalloc(&deviceSrcData, srcN);
		nvtxRangePop();//Malloc

		get_nvtAttrib("Memcpy", 0xFFFF0000);
		hipMemcpy(deviceSrcData, input + src_begin, srcN, hipMemcpyHostToDevice);
		nvtxRangePop();//Memcpy
		
		nvtxRangePop();//Pre-K

		get_nvtAttrib("Kern", 0xFF00FF00);
		fGaussianFilterKernel << <blocks, threadsPerBlock >> > (deviceDestData, deviceSrcData, deviceFilter, FILTER_SIZE, srcRows, srcCols, src_begin, pix_begin);
		hipDeviceSynchronize();
		nvtxRangePop();//Kern

		get_nvtAttrib("Post-K", 0xFF0000FF);
		get_nvtAttrib("Memcpy", 0xFF0000FF);
		hipMemcpy(output + pix_begin, deviceDestData, destN, hipMemcpyDeviceToHost);
		nvtxRangePop();//Memcpy

		get_nvtAttrib("hipFree", 0xFF0000FF);
		hipFree(deviceDestData);
		hipFree(deviceSrcData);
		nvtxRangePop();//hipFree

		offset += destN;
		remainder -= pixels;
		nvtxRangePop();//Post-K
	}
	nvtxRangePop();//K Blocks
	hipFree(deviceFilter);
	//hipFree(deviceSrcData);

}

void cudaMyConv2(float* temp, float* large, float* small, int tRows, int tCols, int lRows, int lCols, int sRows, int sCols){
	float* deviceTemp;
	float* deviceLarge;
	float* deviceSmall;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int datasize = sizeof(float);

	int i1, j1, i2, j2;
	int stRow, sdRow, stCol, sdCol;
	int mtRow, mdRow, mtCol, mdCol;
	int ks, km, ls, lm;
	int k, l, m, n;

	int remainder = tRows * tCols;
	int pixels = MEM_CAP / (2.1 * datasize) ;

	hipMalloc(&deviceSmall, (sRows * sCols) * datasize);
	hipMemcpy(deviceSmall, small, (sRows * sCols) * datasize, hipMemcpyHostToDevice);

	int rounds = 0;

	while (true){
		if (remainder <= 0){
			break;
		}
		//printf("Remainder: %d, Pixels: %d\n", remainder, pixels);
		int pix_begin = (tRows * tCols) - remainder;
		int pix_end = min(tRows * tCols - 1, pix_begin + pixels - 1);
		i1 = pix_begin / tCols;
		j1 = pix_begin % tCols;
		i2 = pix_end / tCols;
		j2 = pix_end % tCols;

		stRow = i1 - sRows + 1;
		sdRow = max(0, stRow);
		stCol = j1 - sCols + 1;
		sdCol = max(0, stCol);
		mtRow = stRow + sRows;
		mdRow = min(lRows, stRow + sRows);
		mtCol = stCol + sCols;
		mdCol = min(lCols, stCol + sCols);
		ks = sdRow - stRow; km = sRows - (mtRow - mdRow);
		ls = sdCol - stCol; lm = sCols - (mtCol - mdCol);
		int m_begin = sdRow;
		int n_begin = sdCol;
		int lrg_begin = m_begin * lCols + n_begin;

		stRow = i2 - sRows + 1;
		sdRow = max(0, stRow);
		stCol = j2 - sCols + 1;
		sdCol = max(0, stCol);
		mtRow = stRow + sRows;
		mdRow = min(lRows, stRow + sRows);
		mtCol = stCol + sCols;
		mdCol = min(lCols, stCol + sCols);
		ks = sdRow - stRow; km = sRows - (mtRow - mdRow);
		ls = sdCol - stCol; lm = sCols - (mtCol - mdCol);
		int m_end = sdRow + (km - ks - 1);
		int n_end = sdCol + (lm - ls - 1);
		int lrg_end = m_end * lCols + n_end;

		int tmpN = (pix_end - pix_begin + 1) * datasize;
		int lrgN = (lrg_end - lrg_begin + 1) * datasize;

		hipMalloc(&deviceTemp, tmpN);
		hipMalloc(&deviceLarge, lrgN);

		blocks = (tmpN / datasize + threadsPerBlock - 1) / threadsPerBlock;
		if (blocks == 0) { blocks = 1; }

		/*if (lrg_begin > 300000){
			hipFree(deviceTemp);
			hipFree(deviceLarge);
			break;
		}*/

		//printf("    tmp_begin: %d, tmp_end: %d, tmp_limit: %d, tmpN: %d\n", pix_begin, pix_end, tRows * tCols, tmpN);
		//printf("    lrg_begin: %d, lrg_end: %d, lrg_limit: %d, lrgN: %d\n", lrg_begin, lrg_end, lRows * lCols, lrgN);
		hipMemcpy(deviceLarge, large + lrg_begin, lrgN, hipMemcpyHostToDevice);
		myConv2Kernel << <blocks, threadsPerBlock >> > (deviceTemp, deviceLarge, deviceSmall, tRows, tCols, lRows, lCols, sRows, sCols, pix_begin, lrg_begin);
		

		hipMemcpy(temp + pix_begin, deviceTemp, tmpN, hipMemcpyDeviceToHost);
		//temp[pix_begin] = 255;
		//temp[pix_end] = 255;
		remainder -= pixels;

		hipDeviceSynchronize();
		hipFree(deviceTemp);
		hipFree(deviceLarge);
	}

	hipFree(deviceSmall);
}

void cudaSobelFilter(unsigned char* input, unsigned char* output, int srcRows, int srcCols){
	get_nvtAttrib("Setup Inner", 0xFF000088);
	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	short* deviceTempData;
	int* deviceSobel_x;
	int* deviceSobel_y;
	int srcN = 3 * srcRows * srcCols;
	double* deviceRangeMin;
	double* deviceRangeMax;
	double rangeMin[1] = { 0.0 };
	double rangeMax[1] = { 0.0 };
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;

	int sobel_x[9], sobel_y[9];

	sobel_x[0] = -1; sobel_x[1] = 0; sobel_x[2] = 1;
	sobel_x[3] = -2; sobel_x[4] = 0; sobel_x[5] = 2;
	sobel_x[6] = -1; sobel_x[7] = 0; sobel_x[8] = 1;

	sobel_y[0] = -1; sobel_y[1] = -2; sobel_y[2] = -1;
	sobel_y[3] = 0; sobel_y[4] = 0; sobel_y[5] = 0;
	sobel_y[6] = 1; sobel_y[7] = 2; sobel_y[8] = 1;

	//int threadsPerBlock = 512;
	//blocks = (srcN + threadsPerBlock - 1) / threadsPerBlock;
	//printf("Blocks: %d\n", blocks);

	hipMalloc(&deviceSrcData, srcN*sizeof(unsigned char));
	hipMalloc(&deviceSobel_x, 9 * sizeof(int));
	hipMalloc(&deviceSobel_y, 9 * sizeof(int));
	hipMalloc(&deviceRangeMin, sizeof(double));
	hipMalloc(&deviceRangeMax, sizeof(double));

	hipMemcpy(deviceSrcData, input, srcN*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(deviceSobel_x, sobel_x, 9 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceSobel_y, sobel_y, 9 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceRangeMin, rangeMin, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(deviceRangeMax, rangeMax, sizeof(double), hipMemcpyHostToDevice);

	chunkRows = IMG_CHUNK / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}
	int rounds = ceil(srcRows / (double)chunkRows);

	short* temp_data = new short[3 * srcRows * srcCols];
	nvtxRangePop();

	get_nvtAttrib("Gradient", 0xFF888888);
	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}
		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceTempData, destN*sizeof(short));

		sobelGradientKernel << <blocks, threadsPerBlock >> > (deviceTempData, deviceSrcData, deviceSobel_x, deviceSobel_y, deviceRangeMin, deviceRangeMax, srcRows, srcCols, offset);

		hipMemcpy(temp_data + offset, deviceTempData, destN*sizeof(short), hipMemcpyDeviceToHost);
		hipFree(deviceTempData);

		offset += destN;
	}
	hipFree(deviceSrcData);
	hipFree(deviceSobel_x);
	hipFree(deviceSobel_y);
	hipFree(deviceRangeMin);
	hipFree(deviceRangeMax);
	nvtxRangePop();
	//printf("Works!\n");

	//hipMemcpy(rangeMin, deviceRangeMin, sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(rangeMax, deviceRangeMax, sizeof(double), hipMemcpyDeviceToHost);

	//printf("Host temp data done");

	get_nvtAttrib("Range Find", 0xFF008800);
	for (int i = 0; i < srcRows; i++){
		for (int j = 0; j < srcCols; j++){
			for (int color = 0; color < 3; color++){
				double value = temp_data[3 * (i * srcCols + j) + color];;
				rangeMin[0] = std::fmin(value, rangeMin[0]);
				rangeMax[0] = std::fmax(value, rangeMax[0]);
			}
		}
	}
	nvtxRangePop();

	//printf("Got here!\n");
	//output = (uchar*)temp_data;
	//return;

	//printf("Range Min: %f, Range Max: %f \n", rangeMin[0], rangeMax[0]);

	//blocks = (srcN + threadsPerBlock - 1) / threadsPerBlock;
	offset = 0;

	get_nvtAttrib("Range", 0xFF888888);
	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}
		blocks = (destN + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceTempData, destN*sizeof(short));
		hipMemcpy(deviceTempData, temp_data + offset, destN*sizeof(short), hipMemcpyHostToDevice);

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));
		sobelRangeKernel << <blocks, threadsPerBlock >> >(deviceDestData, deviceTempData, rangeMin[0], rangeMax[0], 20, 60, offset);

		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);

		hipFree(deviceDestData);
		hipFree(deviceTempData);
		offset += destN;
	}
	nvtxRangePop();
	//uchar minThresh = 20;
	//uchar maxThresh = 60;

	/*for (int i = 0; i < srcRows; i++){
	for (int j = 0; j < srcCols; j++){
	for (int color = 0; color < 3; color++){
	int idx = 3 * (i * srcCols + j) + color;

	int value = 255 * ((temp_data[idx] + rangeMin[0])/(rangeMax[0] - rangeMin[0]));
	if (value >= maxThresh){
	value = 255;
	}
	else if (value < minThresh){
	value = 0;
	}
	output[idx] = value;
	}
	}
	}*/


	//hipFree(deviceTempData);
	//hipFree(deviceDestData);

	delete[] temp_data;
}

void cudaKMeans(unsigned char* input, unsigned char* output, int srcRows, int srcCols, int k_means){
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;

	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	float* device_k_colors;
	int* device_k_count;
	//int* device_hits;
	unsigned char* device_k_index;
	bool* device_convergence;

	float* k_colors = new float[k_means * 3];
	unsigned char* k_index = new unsigned char[srcRows * srcCols];
	int* k_count = new int[k_means];
	//* hits = new int[k_means];

	int srcN = srcRows * srcCols * 3;

	for (int pix = 0; pix < k_means; pix++){
		int i = rand() % srcRows;
		int j = rand() % srcCols;
		for (int color = 0; color < 3; color++){
			k_colors[3 * pix + color] = input[3 * (i * srcCols + j) + color];
		}
	}
	hipMalloc(&device_k_colors, (3 * k_means)*sizeof(float));
	hipMemcpy(device_k_colors, k_colors, 3 * k_means *sizeof(float), hipMemcpyHostToDevice);

	//printf("=== START ===\n");
	//for (int group = 0; group < k_means; group++){
	//printf("Color Group %d: R=%f, G=%f, B=%f \n", group + 1, k_colors[3 * group + 2], k_colors[3 * group + 1], k_colors[3 * group]);
	//}

	bool convergence[1] = { false };

	for (int k = 0; k < srcRows * srcCols; k++){
		k_index[k] = 0;
	}

	chunkRows = (IMG_CHUNK * 0.5) / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}

	hipMalloc(&device_k_count, (k_means)*sizeof(int));
	//hipMalloc(&device_hits, (k_means)*sizeof(int));
	//hipMalloc(&device_k_colors, (3 * k_means)*sizeof(float));
	hipMalloc(&device_convergence, sizeof(bool));

	//hipMalloc(&device_k_index, (srcRows * srcCols)*sizeof(uchar));
	//hipMemcpy(device_k_index, k_index, srcRows * srcCols *sizeof(uchar), hipMemcpyHostToDevice);
	int count = 0;

	while (!convergence[0]){
		//convergence[0] = true; //UNDO
		hipMemcpy(device_convergence, convergence, sizeof(bool), hipMemcpyHostToDevice);
		for (int k = 0; k < k_means; k++){
			k_count[k] = 0;
			//hits[k] = 0;
		}
		hipMemcpy(device_k_count, k_count, k_means * sizeof(int), hipMemcpyHostToDevice);
		//hipMemcpy(device_hits, hits, k_means * sizeof(int), hipMemcpyHostToDevice);
		//printf("Count: %d\n",count);

		int rounds = ceil(srcRows / (float)chunkRows);

		offset = 0;
		for (int step = 0; step < rounds; step++){
			int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
			if (destN <= 0){
				break;
			}
			blocks = ((destN / 3) + threadsPerBlock - 1) / threadsPerBlock;

			hipMalloc(&deviceSrcData, destN*sizeof(unsigned char));
			hipMemcpy(deviceSrcData, input + offset, destN*sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMalloc(&device_k_index, destN*sizeof(unsigned char) / 3);
			hipMemcpy(device_k_index, k_index + (offset / 3), destN*sizeof(unsigned char) / 3, hipMemcpyHostToDevice);

			//kernel
			//kMeansCountingKernel << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_hits, device_k_colors, device_convergence, k_means, srcRows, srcCols,count);
			kMeansCountingKernel << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_k_colors, device_convergence, k_means, srcRows, srcCols);

			hipMemcpy(k_index + (offset / 3), device_k_index, destN*sizeof(unsigned char) / 3, hipMemcpyDeviceToHost);

			hipFree(deviceSrcData);
			hipFree(device_k_index);
			offset += destN;
		}
		hipMemcpy(k_count, device_k_count, (k_means)*sizeof(int), hipMemcpyDeviceToHost);
		//hipMemcpy(hits, device_hits, (k_means)*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(convergence, device_convergence, sizeof(bool), hipMemcpyDeviceToHost);

		//printf("Group Count, step %d::\n",count);
		//for (int i = 0; i < k_means; i++){
		//	printf("Group %d: %d\n",i,k_count[i]);
		//}

		//convergence[0] = true;	//Stopper

		//if (convergence[0])		//RE-ENABLE WHEN ACTUALLY USING
		//break;

		if (count == 200){
			//printf("Stopped at %d!\n",count);
			break;
		}
		count++;

		for (int k = 0; k < 3 * k_means; k++){
			k_colors[k] = 0;
		}
		hipMemcpy(device_k_colors, k_colors, 3 * k_means *sizeof(float), hipMemcpyHostToDevice);

		rounds = ceil(srcRows / (float)chunkRows);

		offset = 0;
		for (int step = 0; step < rounds; step++){
			int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
			if (destN <= 0){
				break;
			}
			blocks = ((destN / 3) + threadsPerBlock - 1) / threadsPerBlock;

			hipMalloc(&deviceSrcData, destN*sizeof(unsigned char));
			hipMemcpy(deviceSrcData, input + offset, destN*sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMalloc(&device_k_index, destN*sizeof(unsigned char) / 3);
			hipMemcpy(device_k_index, k_index + (offset / 3), destN*sizeof(unsigned char) / 3, hipMemcpyHostToDevice);

			kMeansGroupAdjustKernel << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_k_colors, k_means, srcRows, srcCols);

			hipFree(deviceSrcData);
			hipFree(device_k_index);

			offset += destN;
		}
		hipMemcpy(k_colors, device_k_colors, 3 * k_means * sizeof(float), hipMemcpyDeviceToHost);
		//kernel
	}

	int rounds = ceil(srcRows / (float)chunkRows);
	offset = 0;
	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}
		blocks = ((destN / 3) + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));
		hipMalloc(&device_k_index, destN*sizeof(unsigned char) / 3);
		hipMemcpy(device_k_index, k_index + (offset / 3), destN*sizeof(unsigned char) / 3, hipMemcpyHostToDevice);

		//kernel
		kMeansOutputKernel << <blocks, threadsPerBlock >> > (deviceDestData, device_k_index, device_k_colors, srcRows, srcCols);
		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);

		hipFree(deviceDestData);
		hipFree(device_k_index);

		offset += destN;
	}

	//printf("Count: %d\n", count);

	hipFree(device_k_colors);
	hipFree(device_k_count);
	hipFree(device_convergence);

	delete[] k_colors;
	delete[] k_index;
	delete[] k_count;

}

void cudaKMeansFixed(unsigned char* input, unsigned char* output, int srcRows, int srcCols, int k_means){
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks = 0;
	int chunkRows = 0;
	int offset = 0;

	unsigned char* deviceSrcData;
	unsigned char* deviceDestData;
	int* device_k_colors;
	int* device_k_count;
	//int* device_hits;
	unsigned char* device_k_index;
	bool* device_convergence;

	//float* k_colors = new float[k_means * 3];
	int* k_colors = new int[k_means * 3];
	unsigned char* k_index = new unsigned char[srcRows * srcCols];
	int* k_count = new int[k_means];
	//* hits = new int[k_means];

	int srcN = srcRows * srcCols * 3;

	for (int pix = 0; pix < k_means; pix++){
		int i = rand() % srcRows;
		int j = rand() % srcCols;
		for (int color = 0; color < 3; color++){
			k_colors[3 * pix + color] = input[3 * (i * srcCols + j) + color] * ONE;
		}
	}
	hipMalloc(&device_k_colors, (3 * k_means)*sizeof(int));
	hipMemcpy(device_k_colors, k_colors, 3 * k_means *sizeof(int), hipMemcpyHostToDevice);

	//printf("=== START ===\n");
	//for (int group = 0; group < k_means; group++){
	//printf("Color Group %d: R=%f, G=%f, B=%f \n", group + 1, k_colors[3 * group + 2], k_colors[3 * group + 1], k_colors[3 * group]);
	//}

	bool convergence[1] = { false };

	for (int k = 0; k < srcRows * srcCols; k++){
		k_index[k] = 0;
	}

	chunkRows = (IMG_CHUNK * 0.5) / srcCols;
	if (chunkRows == 0){
		chunkRows = srcRows;
	}

	hipMalloc(&device_k_count, (k_means)*sizeof(int));
	//hipMalloc(&device_hits, (k_means)*sizeof(int));
	//hipMalloc(&device_k_colors, (3 * k_means)*sizeof(float));
	hipMalloc(&device_convergence, sizeof(bool));

	//hipMalloc(&device_k_index, (srcRows * srcCols)*sizeof(uchar));
	//hipMemcpy(device_k_index, k_index, srcRows * srcCols *sizeof(uchar), hipMemcpyHostToDevice);
	int count = 0;

	while (!convergence[0]){
		//convergence[0] = true; //UNDO
		hipMemcpy(device_convergence, convergence, sizeof(bool), hipMemcpyHostToDevice);
		for (int k = 0; k < k_means; k++){
			k_count[k] = 0;
			//hits[k] = 0;
		}
		hipMemcpy(device_k_count, k_count, k_means * sizeof(int), hipMemcpyHostToDevice);
		//hipMemcpy(device_hits, hits, k_means * sizeof(int), hipMemcpyHostToDevice);
		//printf("Count: %d\n",count);

		int rounds = ceil(srcRows / (float)chunkRows);

		offset = 0;
		for (int step = 0; step < rounds; step++){
			int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
			if (destN <= 0){
				break;
			}
			blocks = ((destN / 3) + threadsPerBlock - 1) / threadsPerBlock;

			hipMalloc(&deviceSrcData, destN*sizeof(unsigned char));
			hipMemcpy(deviceSrcData, input + offset, destN*sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMalloc(&device_k_index, destN*sizeof(unsigned char) / 3);
			hipMemcpy(device_k_index, k_index + (offset / 3), destN*sizeof(unsigned char) / 3, hipMemcpyHostToDevice);

			//kernel
			//kMeansCountingKernel << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_hits, device_k_colors, device_convergence, k_means, srcRows, srcCols,count);
			kMeansCountingKernelFixed << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_k_colors, device_convergence, k_means, srcRows, srcCols);

			hipMemcpy(k_index + (offset / 3), device_k_index, destN*sizeof(unsigned char) / 3, hipMemcpyDeviceToHost);

			hipFree(deviceSrcData);
			hipFree(device_k_index);
			offset += destN;
		}
		hipMemcpy(k_count, device_k_count, (k_means)*sizeof(int), hipMemcpyDeviceToHost);
		//hipMemcpy(hits, device_hits, (k_means)*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(convergence, device_convergence, sizeof(bool), hipMemcpyDeviceToHost);

		//printf("Group Count, step %d::\n",count);
		//for (int i = 0; i < k_means; i++){
		//	printf("Group %d: %d\n",i,k_count[i]);
		//}

		//convergence[0] = true;	//Stopper

		//if (convergence[0])		//RE-ENABLE WHEN ACTUALLY USING
		//break;

		if (count == 200){
			//printf("Stopped at %d!\n",count);
			break;
		}
		count++;

		for (int k = 0; k < 3 * k_means; k++){
			k_colors[k] = 0;
		}
		hipMemcpy(device_k_colors, k_colors, 3 * k_means *sizeof(int), hipMemcpyHostToDevice);

		rounds = ceil(srcRows / (float)chunkRows);

		offset = 0;
		for (int step = 0; step < rounds; step++){
			int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
			if (destN <= 0){
				break;
			}
			blocks = ((destN / 3) + threadsPerBlock - 1) / threadsPerBlock;

			hipMalloc(&deviceSrcData, destN*sizeof(unsigned char));
			hipMemcpy(deviceSrcData, input + offset, destN*sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMalloc(&device_k_index, destN*sizeof(unsigned char) / 3);
			hipMemcpy(device_k_index, k_index + (offset / 3), destN*sizeof(unsigned char) / 3, hipMemcpyHostToDevice);

			kMeansGroupAdjustKernelFixed << <blocks, threadsPerBlock >> > (deviceSrcData, device_k_index, device_k_count, device_k_colors, k_means, srcRows, srcCols);

			hipFree(deviceSrcData);
			hipFree(device_k_index);

			offset += destN;
		}
		hipMemcpy(k_colors, device_k_colors, 3 * k_means * sizeof(int), hipMemcpyDeviceToHost);
		//kernel
	}

	int rounds = ceil(srcRows / (float)chunkRows);
	offset = 0;
	for (int step = 0; step < rounds; step++){
		int destN = fmin(3 * chunkRows * srcCols, 3 * srcRows * srcCols - offset);
		if (destN <= 0){
			break;
		}
		blocks = ((destN / 3) + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceDestData, destN*sizeof(unsigned char));
		hipMalloc(&device_k_index, destN*sizeof(unsigned char) / 3);
		hipMemcpy(device_k_index, k_index + (offset / 3), destN*sizeof(unsigned char) / 3, hipMemcpyHostToDevice);

		//kernel
		kMeansOutputKernelFixed << <blocks, threadsPerBlock >> > (deviceDestData, device_k_index, device_k_colors, srcRows, srcCols);
		hipMemcpy(output + offset, deviceDestData, destN*sizeof(unsigned char), hipMemcpyDeviceToHost);

		hipFree(deviceDestData);
		hipFree(device_k_index);

		offset += destN;
	}

	//printf("Count: %d\n", count);

	hipFree(device_k_colors);
	hipFree(device_k_count);
	hipFree(device_convergence);

	delete[] k_colors;
	delete[] k_index;
	delete[] k_count;

}

void cudaMySiftDOG(float* current, float* next, float* dog, int curRows, int curCols){
	float* deviceCurrData;
	float* deviceNextData;
	float* deviceDogData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int datasize = sizeof(float);

	int pixels = MEM_CAP / (3 * datasize);

	int remainder = curRows * curCols;

	while (remainder > 0){

		int pix_begin = (curRows * curCols) - remainder;
		int pix_end = min(curRows * curCols - 1, pix_begin + pixels - 1);

		int destN = (pix_end - pix_begin + 1) * datasize;
		int blocks = ((destN / datasize) + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceCurrData, destN);
		hipMalloc(&deviceNextData, destN);
		hipMalloc(&deviceDogData, destN);
		hipMemcpy(deviceCurrData, current + pix_begin, destN, hipMemcpyHostToDevice);
		hipMemcpy(deviceNextData, next + pix_begin, destN, hipMemcpyHostToDevice);

		mySiftDOGKernel << <blocks, threadsPerBlock >> >(deviceCurrData, deviceNextData, deviceDogData);

		hipMemcpy(dog + pix_begin, deviceDogData, destN, hipMemcpyDeviceToHost);

		hipFree(deviceCurrData);
		hipFree(deviceNextData);
		hipFree(deviceDogData);

		remainder -= pixels;
	}

}

void cudaMySiftKeypoints(float* prev_data, float* curr_data, float* next_data, char* answers, unsigned int* key_str, int curRows, int curCols, int key_str_size){
	float* devicePrevData;
	float* deviceCurrData;
	float* deviceNextData;
	char*  deviceAnswers;
	unsigned int* deviceKeyStr;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int datasize = sizeof(float);
	int keysize = sizeof(unsigned int);
	int keybits = keysize * 8;

	int pixels = (MEM_CAP) / (3.5 * datasize);
	pixels -= ceil(pixels / (float)keybits);
	//printf("Pixels: %d\n", pixels);

	int remainder = curRows * curCols;

	while (remainder > 0){

		int pix_begin = (curRows * curCols) - remainder;
		int pix_end = min(curRows * curCols - 1, pix_begin + pixels - 1);

		int block_begin = (pix_begin) / keybits;
		//int block_end = min(key_str_size - 1, (pix_end / keysize));
		int block_end = (pix_end) / keybits;

		int src_begin = max(0, pix_begin - curCols - 1);
		int src_end = min(curRows * curCols - 1, pix_end + curCols + 1);

		int pixN = (pix_end - pix_begin + 1) * datasize;
		int ansN = (pix_end - pix_begin + 1);
		int srcN = (src_end - src_begin + 1) * datasize;
		int blocks = ((pixN / datasize) + threadsPerBlock - 1) / threadsPerBlock;
		int strN = (block_end - block_begin + 1) * keysize;

		hipMalloc(&devicePrevData, srcN);
		hipMalloc(&deviceCurrData, srcN);
		hipMalloc(&deviceNextData, srcN);
		hipMalloc(&deviceAnswers, ansN);
		//hipMalloc(&deviceKeyStr, strN);
		hipMemcpy(devicePrevData, prev_data + src_begin, srcN, hipMemcpyHostToDevice);
		hipMemcpy(deviceCurrData, curr_data + src_begin, srcN, hipMemcpyHostToDevice);
		hipMemcpy(deviceNextData, next_data + src_begin, srcN, hipMemcpyHostToDevice);
		//hipMemcpy(deviceKeyStr, key_str + block_begin, strN, hipMemcpyHostToDevice);

		mySiftKeypointsKernel << <blocks, threadsPerBlock >> >(devicePrevData, deviceCurrData, deviceNextData, deviceAnswers, curRows, curCols, pix_begin, src_begin, block_begin, keybits);

		//hipMemcpy(key_str + block_begin, deviceKeyStr, strN, hipMemcpyDeviceToHost);
		hipMemcpy(answers + pix_begin, deviceAnswers, ansN, hipMemcpyDeviceToHost);

		hipFree(devicePrevData);
		hipFree(deviceCurrData);
		hipFree(deviceNextData);
		hipFree(deviceAnswers);
		//hipFree(deviceKeyStr);

		remainder -= pixels;
	}
}

/*void cudaMySiftOrMagGen(float* curr_data, float* or_mag, int curRows, int curCols){
	float* deviceCurrData;
	float* deviceOrMag;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int datasize = sizeof(float);

	int offset = 0;
	int remainder = curRows * curCols;

	int pixels = (MEM_CAP) / (3.2 * datasize);
	//printf("Pixels: %d\n", pixels);

	while (remainder > 0){

		int pix_begin = (curRows * curCols - remainder);
		int pix_end = min(curRows * curCols - 1, pix_begin + pixels - 1);
		int pixN = (pix_end - pix_begin + 1) * datasize;

		int src_begin = max(0, pix_begin - curCols - 1);
		int src_end = min(curRows * curCols - 1, pix_end + curCols + 1);
		//printf("src_begin: %d, src_end: %d\n", src_begin, src_end);
		int srcN = (src_end - src_begin + 1) * datasize;
		int blocks = ((pixN / datasize) + threadsPerBlock - 1) / threadsPerBlock;

		//printf("pixN: %d, srcN: %d\n", pixN, srcN);

		//hipMalloc(&deviceCurrData, srcN);
		//hipMalloc(&deviceOrMag, pixN);
		//printf("Test: %d\n", curr_data);
		//hipMemcpy(deviceCurrData, curr_data + src_begin, srcN, hipMemcpyHostToDevice);

		//printf("pix_begin: %d, pix_end: %d, pixN: %d, blocks: %d\n", pix_begin, pix_end, pixN, blocks);
		//printf("Blocks: %d, ThreadsPerBlock: %d\n", blocks, threadsPerBlock);

		//mySiftOrMagKernel << <blocks, threadsPerBlock >> >(NULL, NULL, curRows, curCols, pix_begin, pix_end, src_begin, src_end);
		testKernel << <blocks, threadsPerBlock >> >(NULL);
		
		//printf("  Threads\n");
		//hipMemcpy(or_mag + (pix_begin * 2), deviceOrMag, pixN, hipMemcpyDeviceToHost);
		//printf("  Threads2\n");

		//hipFree(deviceOrMag);
		//hipFree(deviceCurrData);

		offset += pixels;
		remainder -= pixels;
	}
}*/

void cudaTest(int curRows, int curCols){
	//hipDeviceSynchronize();
	//float* deviceCurrData;
	//float* deviceOrMag;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int datasize = sizeof(float);

	int remainder = curRows * curCols;

	int pixels = MEM_CAP / (3.5 * datasize);
	int blocks = 1;
	testKernel <<<20, 256 >>>();
	hipError_t err = hipGetLastError();
	if (hipSuccess != err || true){
		printf("Fish %s\n", hipGetErrorString(err));
	}
	gpuErrchk(hipDeviceSynchronize());

	remainder -= pixels;
}

void cudaMySiftOrMagGen(float* curr_data, float* or_mag, int curRows, int curCols){
	float* deviceCurrData;
	float* deviceOrMag;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int datasize = sizeof(float);

	int offset = 0;
	int remainder = curRows * curCols;

	int pixels = (MEM_CAP) / (3.2 * datasize);

	while (remainder > 0){

		int pix_begin = (curRows * curCols - remainder);
		int pix_end = min(curRows * curCols - 1, pix_begin + pixels - 1);
		int pixN = (pix_end - pix_begin + 1) * 2 * datasize;

		int src_begin = max(0, pix_begin - curCols - 1);
		int src_end = min(curRows * curCols - 1, pix_end + curCols + 1);
		int srcN = (src_end - src_begin + 1) * datasize;
		int blocks = ((pixN / (2 * datasize)) + threadsPerBlock - 1) / threadsPerBlock;

		hipMalloc(&deviceCurrData, srcN);
		hipMalloc(&deviceOrMag, pixN);
		hipMemcpy(deviceCurrData, curr_data + src_begin, srcN, hipMemcpyHostToDevice);

		mySiftOrMagKernel << <blocks, threadsPerBlock >> >(deviceCurrData, deviceOrMag, curRows, curCols, pix_begin, pix_end, src_begin, src_end);
		//printf("  Fish\n");
		hipMemcpy(or_mag + pix_begin * 2, deviceOrMag, pixN, hipMemcpyDeviceToHost);

		hipFree(deviceCurrData);
		hipFree(deviceOrMag);

		offset += pixels;
		remainder -= pixels;
	}
}

void cudaMySiftCountSort(unsigned int* data, unsigned int* index, int d, int exp){
	//printf("Start Count Sort\n");
	get_nvtAttrib("Count Sort", 0xFF000088);
	unsigned int* deviceData;
	unsigned int* deviceOutData;
	unsigned int* deviceIndex;
	unsigned int* deviceOutIndex;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int srcN = d * sizeof(unsigned int);
	int blocks = (d + threadsPerBlock - 1) / threadsPerBlock;
	//int exp_cur = 1;

	int* count = new int[10];
	for (int i = 0; i < 10; i++) count[i] = 0;
	int* deviceCount;
	hipMalloc((void**)&deviceData, srcN);
	hipMalloc((void**)&deviceOutData, srcN);
	hipMalloc((void**)&deviceIndex, srcN);
	hipMalloc((void**)&deviceOutIndex, srcN);
	hipMalloc((void**)&deviceCount, 10 * sizeof(int));
	//printf("Test\n");

	//gpuErrchk(hipMemcpy(deviceData, data, srcN, hipMemcpyHostToDevice));
	//gpuErrchk(hipMemcpy(deviceIndex, index, srcN, hipMemcpyHostToDevice));
	//gpuErrchk(hipMemcpy(deviceCount, count, 10 * sizeof(int), hipMemcpyHostToDevice));
	//gpuErrchk(hipPeekAtLastError());

	

	hipMemcpy(deviceData, data, srcN, hipMemcpyHostToDevice);
	hipMemcpy(deviceIndex, index, srcN, hipMemcpyHostToDevice);
	hipMemcpy(deviceCount, count, 10 * sizeof(int), hipMemcpyHostToDevice);

	//printf("Setup\n");

	int exp_curr = 1;
	while (exp / exp_curr > 0){
		//printf("Loop: %d\n", exp_curr);
		//printf("Memcpy In: %d\n", exp_curr);

		mySiftCountingKernel << <blocks, threadsPerBlock >> > (deviceData, deviceCount, exp_curr, d);
		//gpuErrchk(hipPeekAtLastError());
		hipDeviceSynchronize();
		
		hipMemcpy(count, deviceCount, 10 * sizeof(int), hipMemcpyDeviceToHost);
		//printf("Memcpy Out: %d\n", exp_curr);

		for (int i = 1; i < 10; i++){
			count[i] += count[i - 1];
		}
		hipMemcpy(deviceCount, count, 10 * sizeof(int), hipMemcpyHostToDevice);

		mySiftCountSortKernel << <blocks, threadsPerBlock >> > (deviceData, deviceOutData, deviceIndex, deviceOutIndex, deviceCount, exp_curr, d);
		hipDeviceSynchronize();
		mySiftCountSortSwitchKernel << <blocks, threadsPerBlock >> > (deviceData, deviceOutData, deviceIndex, deviceOutIndex, deviceCount, exp_curr, d);

		//printf("Kernels: %d\n", exp_curr);

		
		for (int i = 0; i < 10; i++) count[i] = 0;
		exp_curr *= 10;
	}

	hipMemcpy(data, deviceOutData, srcN, hipMemcpyDeviceToHost);
	hipMemcpy(index, deviceOutIndex, srcN, hipMemcpyDeviceToHost);

	hipFree(deviceData);
	hipFree(deviceOutData);
	hipFree(deviceIndex);
	hipFree(deviceOutIndex);
	hipFree(deviceCount);

	delete[] count;
	//mySiftCountSortKernel << <blocks, threadsPerBlock >> > (deviceData, deviceOutData, deviceIndex, deviceOutIndex, deviceCount, exp);
	nvtxRangePop();

}

#endif